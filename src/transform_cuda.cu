#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <bits/huge_val.h>

namespace {

__global__ void kernel(const int count, double* x, double* y) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < count) {
        
        i += gridDim.x * blockDim.x;
    }
}

}  // anonymous namespace

namespace cu {

void func(const int grid_size, const int block_size, const int count,
                    double* x, double* y) {}
                        
void func1(int* a) {
    *a = 99;
}

}  // namespace cu